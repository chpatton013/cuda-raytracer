#include "hip/hip_runtime.h"
#include "Util.h"

#include <math.h>
#include <stdint.h>

/**
 * This disgusting function returns a very good approximation of the inverse
 * square root.
 *
 * This function is taken from the Quake III Arena source code.
 * Though the code was edited for style, the content and comments remain
 * unchanged.
 */
float Q_rsqrt(float number) {
   long i;
   float x2, y;
   float threehalfs = 1.5f;

   x2 = number * 0.5f;
   y = number;
   i = *((long*)&y);                      // evil floating point bit level hacking
   i = 0x5f3759df - ( i >> 1 );           // what the fuck?
   y = *((float*)&i);
   y = y * (threehalfs - (x2 * y * y));   // 1st iteration
   y = y * (threehalfs - (x2 * y * y));   // 2nd iteration, this can be removed

   return y;
}

void zero_vector(float* v) {
   for (int ndx = 0; ndx < 3; ++ndx) {
      v[ndx] = 0.0f;
   }
}
void copy(float* v_src, float* v_dest) {
   for (int ndx = 0; ndx < 3; ++ndx) {
      v_dest[ndx] = v_src[ndx];
   }
}
float dot(float* v1, float* v2) {
   float result = 0.0f;

   for (int ndx = 0; ndx < 3; ++ndx) {
      result += v1[ndx] * v2[ndx];
   }

   return result;
}
void cross(float* v1, float* v2, float* v_dest) {
   // Sarrus' Rule for 3x3 determinants
   v_dest[0] = v1[1] * v2[2] - v1[2] * v2[1];
   v_dest[1] = v1[2] * v2[0] - v1[0] * v2[2];
   v_dest[2] = v1[0] * v2[1] - v1[1] * v2[0];
}
float mag(float* v) {
   return 1 / Q_rsqrt(dot(v, v));
}
void norm(float* v_src, float* v_dest) {
   scale(v_src, 1.0f / mag(v_src), v_dest);
}
void norm_i(float* v) {
   scale_i(v, 1.0f / mag(v));
}
void add(float* v1, float* v2, float* v_dest) {
   for (int ndx = 0; ndx < 3; ++ndx) {
      v_dest[ndx] = v1[ndx] + v2[ndx];
   }
}
void add_i(float* v1, float* v2) {
   for (int ndx = 0; ndx < 3; ++ndx) {
      v1[ndx] += v2[ndx];
   }
}
void sub(float* v1, float* v2, float* v_dest) {
   for (int ndx = 0; ndx < 3; ++ndx) {
      v_dest[ndx] = v1[ndx] - v2[ndx];
   }
}
void sub_i(float* v1, float* v2) {
   for (int ndx = 0; ndx < 3; ++ndx) {
      v1[ndx] -= v2[ndx];
   }
}
void scale(float* v_src, float s, float* v_dest) {
   for (int ndx = 0; ndx < 3; ++ndx) {
      v_dest[ndx] = v_src[ndx] * s;
   }
}
void scale_i(float* v, float s) {
   for (int ndx = 0; ndx < 3; ++ndx) {
      v[ndx] *= s;
   }
}
void scalev(float* v1, float* v2, float* v_dest) {
   for (int ndx = 0; ndx < 3; ++ndx) {
      v_dest[ndx] = v1[ndx] * v2[ndx];
   }
}
void scalev_i(float* v1, float* v2) {
   for (int ndx = 0; ndx < 3; ++ndx) {
      v1[ndx] *= v2[ndx];
   }
}
