
#include <hip/hip_runtime.h>
#include <stdio.h>


static void HandleError( hipError_t err,
    const char *file,
    int line ) {
  if (err != hipSuccess) {
    printf( "%s in %s at line %d\n", hipGetErrorString( err ),
        file, line );
    exit( EXIT_FAILURE );
  }
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__global__ void MMKernel(int i)
{
  printf("world\n");
  
  printf("world\n");
}

void cudaWrapper()
{

  int c;
  int *dev_c;
  
  HANDLE_ERROR( hipMalloc( (void**)&dev_c, sizeof(int) ) );
  
  MMKernel<<<1,1>>>(1);
  
  
  HANDLE_ERROR( hipMemcpy( &c, dev_c, sizeof(int), hipMemcpyDeviceToHost ) );

  HANDLE_ERROR( hipFree( dev_c ) );  
  
  printf("hello\n");  

}